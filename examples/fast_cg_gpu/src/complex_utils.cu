#include "../includes/complex_utils.h"
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include <hipfft/hipfftXt.h>
#include <hipblas.h>

namespace faster_dft
{

template <>
void complex_gemm(const hipDoubleComplex *A, const hipDoubleComplex *B, hipDoubleComplex *C,
                  const int M, const int K, const int N, hipblasHandle_t handle)
{
    hipDoubleComplex ONE, ZERO;
    ONE.y = ZERO.x = ZERO.y = 0.0;
    ONE.x = 1.0;

    hipblasOperation_t transa = HIPBLAS_OP_N;
    hipblasOperation_t transb = HIPBLAS_OP_N;

    hipblasZgemm(handle, transa, transb, N, M, K, &ONE, B, N, A, K, &ZERO, C, N);

}

template <>
void complex_gemm(const hipComplex *A, const hipComplex *B, hipComplex *C,
                  const int M, const int K, const int N, hipblasHandle_t handle)
{
    hipComplex ONE, ZERO;
    ONE.y = ZERO.x = ZERO.y = 0.0;
    ONE.x = 1.0;

    hipblasOperation_t transa = HIPBLAS_OP_N;
    hipblasOperation_t transb = HIPBLAS_OP_N;

    hipblasCgemm(handle, transa, transb, N, M, K, &ONE, B, N, A, K, &ZERO, C, N);

}

template void complex_gemm(const hipDoubleComplex *A, const hipDoubleComplex *B, hipDoubleComplex *C, const int M, const int K, const int N, hipblasHandle_t handle);
template void complex_gemm(const hipComplex *A, const hipComplex *B, hipComplex *C, const int M, const int K, const int N, hipblasHandle_t handle);
}
